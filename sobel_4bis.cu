#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>


__global__ void grayscale_sobel_shared( unsigned char * rgb, unsigned char * s, std::size_t cols, std::size_t rows , std::size_t rowsOffset) {

    extern __shared__ int shared_g[];

    auto outputBlockDim_x = blockDim.x-2;
    auto outputBlockDim_y = blockDim.y-2;

    auto gId_x = blockIdx.x * outputBlockDim_x + threadIdx.x;//global id x
    auto gId_y = blockIdx.y * outputBlockDim_y + threadIdx.y;//global id y

    auto lId_x = threadIdx.x; //local id x
    auto lId_y = threadIdx.y; //local id y

    if( gId_x < cols && gId_y < rows){
      shared_g[ lId_y * blockDim.x + lId_x ] = (
			    307 * rgb[ 3 * ( (rowsOffset * cols) + gId_y * cols + gId_x ) ]
			    + 604 * rgb[ 3 * ( (rowsOffset * cols) + gId_y * cols + gId_x ) + 1 ]
			    + 113 * rgb[  3 * ( (rowsOffset * cols) + gId_y * cols + gId_x ) + 2 ]
			    ) >> 10;
        //shared_g[ lId_y * blockDim.y + lId_x ] = g[ gId_y * cols + gId_x ];//charger g dans la shared
            
        

        __syncthreads();//attendre que tous les threads aient chargé

        
        //traitement Sobel avec la mémoire partagée
        int h, v, res;
        // GREY = ( 307 * R + 604 * G + 113 * B ) / 1024

        if( lId_x > 0 && lId_x <= outputBlockDim_x && lId_y > 0 && lId_y <= outputBlockDim_y) {
        // Horizontal
            h =     shared_g[ ((lId_y - 1) * blockDim.x + lId_x - 1) ] -     shared_g[ ((lId_y - 1) * blockDim.x + lId_x + 1) ]
                + 2 * shared_g[ ( lId_y      * blockDim.x + lId_x - 1) ] - 2 * shared_g[ ( lId_y      * blockDim.x + lId_x + 1) ]
                +     shared_g[ ((lId_y + 1) * blockDim.x + lId_x - 1) ] -     shared_g[ ((lId_y + 1) * blockDim.x + lId_x + 1) ];

            // Vertical

            v =     shared_g[ ((lId_y - 1) * blockDim.x + lId_x - 1) ] -     shared_g[ ((lId_y + 1) * blockDim.x + lId_x - 1) ]
                + 2 * shared_g[ ((lId_y - 1) * blockDim.x + lId_x    ) ] - 2 * shared_g[ ((lId_y + 1) * blockDim.x + lId_x    ) ]
                +     shared_g[ ((lId_y - 1) * blockDim.x + lId_x + 1) ] -     shared_g[ ((lId_y + 1) * blockDim.x + lId_x + 1) ];

            h = min(h, 255);
            v = min(v, 255);
        
        res = h*h + v*v;
        //verif si > 255*255
        res = min(res, 255*255);

        // s[  ] = sqrt(res);
        s [ (rowsOffset * cols) + (gId_y * cols) + gId_x ] = (int) sqrt((float)res);
        //s [ gId_y * cols + gId_x ] = 127;

        } 
    }   
}


void print_if_err(hipError_t erreur);

int main()
{
  cv::Mat m_in = cv::imread("in2.jpg", cv::IMREAD_UNCHANGED );

  auto rows = m_in.rows;
  auto cols = m_in.cols;

  unsigned char * rgb;
  unsigned char * s;
  unsigned char * rgb_d;
  unsigned char * s_d;

  const int batch = 1;
  int outChunkSize = (rows-1)/batch +1;
  int inChunkSize = outChunkSize + 2;

  hipStream_t streams[batch];

  hipHostMalloc( &rgb, 3 * rows * cols * sizeof(char));
  hipHostMalloc( &s, rows * cols * sizeof(char));
  memcpy(rgb,m_in.data,3 * rows * cols * sizeof(char));

  cv::Mat m_out( rows, cols, CV_8UC1, s );

  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  hipMalloc( &rgb_d, 3 * rows * cols * sizeof(char));
  hipMalloc( &s_d, rows * cols * sizeof(char));

  auto kernelOffset = 0;
  auto memCpyHtDOffset = 0;
  auto memCpyDtHOffset = 1;

  dim3 t_s( 32, 32 );
  dim3 b_s( ( cols - 1) / (t_s.x-2) +1 , (inChunkSize -1) / (t_s.y-2) +1 );
  hipStreamCreate(&streams[0]);
  hipMemcpyAsync(rgb_d + memCpyHtDOffset * cols * 3, rgb + memCpyHtDOffset * cols * 3,  3 * std::min(inChunkSize, rows - memCpyHtDOffset) * cols, hipMemcpyHostToDevice, streams[0]); // this changes between here and for loop
  grayscale_sobel_shared<<< b_s, t_s, (t_s.x)*(t_s.y)*sizeof(int),streams[0]>>>(rgb_d, s_d, cols, std::min(inChunkSize, rows - kernelOffset),kernelOffset);
  hipMemcpyAsync( s + memCpyDtHOffset * cols, s_d + memCpyDtHOffset * cols , std::min(outChunkSize, rows - memCpyDtHOffset) * cols, hipMemcpyDeviceToHost, streams[0] );

  memCpyHtDOffset += inChunkSize; // this changes between here and for loop
  kernelOffset += outChunkSize;
  memCpyDtHOffset += outChunkSize;

  for (int i = 1; i < batch;i++) {
    std::cout << " y : " << kernelOffset << " - "<< std::min(inChunkSize, rows - kernelOffset) << std::endl;
    hipStreamCreate( &streams[ i ]);
    hipMemcpyAsync(rgb_d + memCpyHtDOffset * cols * 3, rgb + memCpyHtDOffset * cols * 3,  3 * std::min(outChunkSize, rows - memCpyHtDOffset) * cols, hipMemcpyHostToDevice, streams[i]);
    grayscale_sobel_shared<<< b_s, t_s, (t_s.x)*(t_s.y)*sizeof(int),streams[i]>>>(rgb_d, s_d, cols, std::min(inChunkSize, rows - kernelOffset),kernelOffset);
    hipMemcpyAsync( s + memCpyDtHOffset * cols, s_d + memCpyDtHOffset * cols , std::min(outChunkSize, rows - memCpyDtHOffset) * cols, hipMemcpyDeviceToHost, streams[i] );

    memCpyHtDOffset += outChunkSize;
    kernelOffset += outChunkSize;
    memCpyDtHOffset += outChunkSize;
  }
  hipError_t erreur_kernel = hipGetLastError();
  print_if_err(erreur_kernel);

  hipDeviceSynchronize();

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  std::cout << "All took " << elapsedTime << "ms" << std::endl;
  hipEventDestroy(start);
  hipEventDestroy(stop);

  cv::imwrite( "out4bis-cu.jpg", m_out );

  for (int i = 0; i < batch;i++) {
    hipStreamDestroy( streams[ i ] );
  }

  hipFree( rgb_d );
  hipFree( s_d );

  return 0;
}

void print_if_err(hipError_t erreur) {
    if(erreur != hipSuccess) {
        std::cout<< "Erreur : " << hipGetErrorString(erreur) <<  std::endl;
        exit(1);
    }
}

