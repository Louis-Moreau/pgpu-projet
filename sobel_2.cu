#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void grayscale( unsigned char * rgb, unsigned char * g, std::size_t cols, std::size_t rows ) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;
  // GREY = ( 307 * R + 604 * G + 113 * B ) / 1024
  if( i < cols && j < rows ) {
    g[ j * cols + i ] = (
			 307 * rgb[ 3 * ( j * cols + i ) ]
			 + 604 * rgb[ 3 * ( j * cols + i ) + 1 ]
			 + 113 * rgb[  3 * ( j * cols + i ) + 2 ]
			 ) >> 10;
  }
}


__global__ void sobel_shared( unsigned char * g, unsigned char * s, std::size_t cols, std::size_t rows ) {

    extern __shared__ int shared_g[]; //mémoire partagée (local pour le bloc multi proc)

    auto outputBlockDim_x = blockDim.x-2;
    auto outputBlockDim_y = blockDim.y-2;

    auto gId_x = blockIdx.x * outputBlockDim_x + threadIdx.x;//id i global pour toute l'image
    auto gId_y = blockIdx.y * outputBlockDim_y  + threadIdx.y;//id j global pour toute l'image

    auto lId_x = threadIdx.x; //id i local au bloc pour la portion d'image du bloc 
    auto lId_y = threadIdx.y; //id j local au bloc pour la portion d'image du bloc

    if( gId_x < cols && gId_y < rows){
        shared_g[ lId_y * blockDim.x + lId_x ] = g[ gId_y * cols + gId_x ];//charger g dans la shared
    

        __syncthreads();//attendre que tous les threads aient chargé

        
        //traitement Sobol avec la mémoire partagée
        int h, v, res;
        // GREY = ( 307 * R + 604 * G + 113 * B ) / 1024

        if( lId_x > 0 && lId_x <= outputBlockDim_x && lId_y > 0 && lId_y <= outputBlockDim_y && gId_y < (rows-1) && gId_x < (cols-1)) {
        // Horizontal
            h =     shared_g[ ((lId_y - 1) * blockDim.x + lId_x - 1) ] -     shared_g[ ((lId_y - 1) * blockDim.x + lId_x + 1) ]
                + 2 * shared_g[ ( lId_y      * blockDim.x + lId_x - 1) ] - 2 * shared_g[ ( lId_y      * blockDim.x + lId_x + 1) ]
                +     shared_g[ ((lId_y + 1) * blockDim.x + lId_x - 1) ] -     shared_g[ ((lId_y + 1) * blockDim.x + lId_x + 1) ];

            // Vertical

            v =     shared_g[ ((lId_y - 1) * blockDim.x + lId_x - 1) ] -     shared_g[ ((lId_y + 1) * blockDim.x + lId_x - 1) ]
                + 2 * shared_g[ ((lId_y - 1) * blockDim.x + lId_x    ) ] - 2 * shared_g[ ((lId_y + 1) * blockDim.x + lId_x    ) ]
                +     shared_g[ ((lId_y - 1) * blockDim.x + lId_x + 1) ] -     shared_g[ ((lId_y + 1) * blockDim.x + lId_x + 1) ];

            h = min(h, 255);
            v = min(v, 255);
        
        res = h*h + v*v;
        //verif si > 255*255
        res = min(res, 255*255);

        // s[  ] = sqrt(res);
        s [ gId_y * cols + gId_x ] = (int) sqrt((float)res);
        //s [ gId_y * cols + gId_x ] = 127;

        } 
    }
}


void print_if_err(hipError_t erreur);

int main(int argc, char** argv)
{

    if(argc != 5){
        std::cout << "Error argument number" << std::endl << "Usage : <image_in> <image_out> <blockDim.x> <blockDim.y>, please verify that <image_in> is jpg and is in ./image/input/" << std::endl;
        exit(1);
      }
      
      const int blockSizeX = atoi(argv[3]);
      const int blockSizeY = atoi(argv[4]);
    
      if( (blockSizeX * blockSizeY) > 1024 ){
        std::cout << "Error block dimension" << std::endl << "<blockDim.x> * <blockDim.y> must be lower or equal to 1024. And both must be positive" << std::endl;
        exit(1);
      }
    
      std::string path_in = argv[1];
      path_in = "./images/input/" + path_in;
    
      std::string path_out = argv[2];
      path_out = "./images/output/" + path_out;


  cv::Mat m_in = cv::imread( path_in, cv::IMREAD_UNCHANGED );

  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;
  
  std::vector< unsigned char > g( rows * cols );
  std::vector< unsigned char > s( rows * cols );
  
  cv::Mat m_out( rows, cols, CV_8UC1, s.data() );
  
  unsigned char * rgb_d;
  unsigned char * g_d;
  unsigned char * s_d;

  hipEvent_t start,stop, startK, stopK;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  
  hipMalloc( &rgb_d, 3 * rows * cols );
  hipMalloc( &g_d, rows * cols );
  hipMalloc( &s_d, rows * cols );

  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );

  dim3 t( blockSizeX, blockSizeY ); //32, 32
  dim3 b( ( cols - 1) / t.x + 1 , ( rows - 1 ) / t.y + 1 );
  
  hipEventCreate(&startK);
  hipEventCreate(&stopK);
  hipEventRecord(startK, 0);
  
  grayscale<<< b, t >>>( rgb_d, g_d, cols, rows );
  //hipMemcpy( g.data(), g_d, rows * cols, hipMemcpyDeviceToHost );
  hipDeviceSynchronize();

  dim3 t_s( blockSizeX, blockSizeY ); //17, 32
  dim3 b_s( ( cols - 1) / (t_s.x-2) +1 , ( rows - 1 ) / (t_s.y-2) +1 );

  sobel_shared<<< b_s, t_s, (t_s.x)*(t_s.y)*sizeof(int)>>>(g_d, s_d, cols, rows);
  hipDeviceSynchronize();

  hipEventRecord(stopK, 0);
  hipEventSynchronize(stopK);
  float elapsedTimeK;
  hipEventElapsedTime(&elapsedTimeK, startK, stopK);
  std::cout << "Kernel took " << elapsedTimeK << "ms" << std::endl;
  hipEventDestroy(startK);
  hipEventDestroy(stopK);




  hipMemcpy( s.data(), s_d, rows * cols, hipMemcpyDeviceToHost );

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  std::cout << "All took " << elapsedTime << "ms" << std::endl;
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipError_t erreur_kernel = hipGetLastError();
  print_if_err(erreur_kernel);
  

  cv::imwrite( path_out, m_out );

  hipFree( rgb_d );
  hipFree( g_d );
  hipFree( s_d );

  return 0;
}

void print_if_err(hipError_t erreur) {
    if(erreur != hipSuccess) {
        std::cout<< "Erreur : " << hipGetErrorString(erreur) <<  std::endl;
        exit(1);
    }
}

