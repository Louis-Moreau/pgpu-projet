#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void grayscale( unsigned char * rgb, unsigned char * g, std::size_t cols, std::size_t rows ) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;
  // GREY = ( 307 * R + 604 * G + 113 * B ) / 1024
  if( i < cols && j < rows ) {
    g[ j * cols + i ] = (
			 307 * rgb[ 3 * ( j * cols + i ) ]
			 + 604 * rgb[ 3 * ( j * cols + i ) + 1 ]
			 + 113 * rgb[  3 * ( j * cols + i ) + 2 ]
			 ) >> 10;
  }
}

__global__ void sobel( unsigned char * g, unsigned char * s, std::size_t cols, std::size_t rows ) {
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    auto j = blockIdx.y * blockDim.y + threadIdx.y;
    int h, v, res;
    // GREY = ( 307 * R + 604 * G + 113 * B ) / 1024
    if( i < cols -1 && i > 0 && j < rows -1 && j > 0) {
       // Horizontal
	    h =     g[((j - 1) * cols + i - 1) ] -     g[((j - 1) * cols + i + 1) ]
            + 2 * g[( j      * cols + i - 1) ] - 2 * g[( j      * cols + i + 1) ]
            +     g[((j + 1) * cols + i - 1) ] -     g[((j + 1) * cols + i + 1) ];

        // Vertical

        v =     g[((j - 1) * cols + i - 1) ] -     g[((j + 1) * cols + i - 1) ]
            + 2 * g[((j - 1) * cols + i    ) ] - 2 * g[((j + 1) * cols + i    ) ]
            +     g[((j - 1) * cols + i + 1) ] -     g[((j + 1) * cols + i + 1) ];

        h = min(h, 255);
        v = min(v, 255);
       
       res = h*h + v*v;
       //verif si > 255*255
       res = min(res, 255*255);

       // s[  ] = sqrt(res);
       s [ j * cols + i ] = (int) sqrt((float)res);

    } 
}

void print_if_err(hipError_t erreur);

int main()
{
  cv::Mat m_in = cv::imread("./images/input/in.jpg", cv::IMREAD_UNCHANGED );

  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;
  
  std::vector< unsigned char > g( rows * cols );
  std::vector< unsigned char > s( rows * cols );
  
  cv::Mat m_out( rows, cols, CV_8UC1, s.data() );
  
  unsigned char * rgb_d;
  unsigned char * g_d;
  unsigned char * s_d;

  hipEvent_t start,stop, startK, stopK;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  
  hipMalloc( &rgb_d, 3 * rows * cols );
  hipMalloc( &g_d, rows * cols );
  hipMalloc( &s_d, rows * cols );

  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );

  dim3 t( 32, 32 );
  dim3 b( ( cols - 1) / t.x + 1 , ( rows - 1 ) / t.y + 1 );
  
  hipEventCreate(&startK);
  hipEventCreate(&stopK);
  hipEventRecord(startK, 0);
  
  grayscale<<< b, t >>>( rgb_d, g_d, cols, rows );
  //hipMemcpy( g.data(), g_d, rows * cols, hipMemcpyDeviceToHost );
  hipDeviceSynchronize();

  sobel<<< b, t >>>(g_d, s_d, cols, rows);
  hipDeviceSynchronize();

  hipEventRecord(stopK, 0);
  hipEventSynchronize(stopK);
  float elapsedTimeK;
  hipEventElapsedTime(&elapsedTimeK, startK, stopK);
  std::cout << "Kernel took " << elapsedTimeK << "ms" << std::endl;
  hipEventDestroy(startK);
  hipEventDestroy(stopK);




  hipMemcpy( s.data(), s_d, rows * cols, hipMemcpyDeviceToHost );

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  std::cout << "All took " << elapsedTime << "ms" << std::endl;
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipError_t erreur_kernel = hipGetLastError();
  print_if_err(erreur_kernel);
  

  cv::imwrite( "./images/output/out-cu.jpg", m_out );

  hipFree( rgb_d );
  hipFree( g_d );
  hipFree( s_d );

  return 0;
}

void print_if_err(hipError_t erreur) {
    if(erreur != hipSuccess) {
        std::cout<< "Erreur : " << hipGetErrorString(erreur) <<  std::endl;
        exit(1);
    }
}

